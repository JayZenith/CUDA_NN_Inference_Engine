#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "kernels.cu"

void forward_gpu(float* d_input, float* d_W1, float* d_b1,
                 float* d_W2, float* d_b2, float* d_output,
                 int input_size, int hidden_size, int output_size) {
    // allocating GPU memory for the hidden layer
    float* d_hidden;
    hipMalloc(&d_hidden, hidden_size * sizeof(float));

    //Linear part (matrix multiply + bias)
    dim3 threads(16,16);
    dim3 blocks((hidden_size+15)/16, (1+15)/16);
    // layer1: input * W1
    //const float* A, const float* B, float* C, int M, int N, int K
    // A = d_input (1, input_size) - row vector
    // B = d_W1 (input_size, hidden_size)
    // C = d_hidden (1, hidden_size)
    // (1 × input_size) × (input_size × hidden_size) = (1 × hidden_size)
    matmul_tiled<<<blocks, threads>>>(d_input, d_W1, d_hidden, 1, input_size, hidden_size);
    hipDeviceSynchronize();

    // Add bias (simple kernel)

    // Apply ReLU activation (non-linear step)
    // ReLU: relu_kernel(float* x, int size)
    relu_kernel<<<(hidden_size+255)/256,256>>>(d_hidden, hidden_size);

    // layer2: hidden * W2
    // output = (1 × hidden_size) × (hidden_size × output_size) = (1 × output_size)
    matmul_tiled<<<dim3((output_size+15)/16,(1+15)/16), dim3(16,16)>>>(d_hidden, d_W2, d_output, 1, hidden_size, output_size);
    hipDeviceSynchronize();

    // Add bias (simple kernel)

    // Classification
    // <<<1,1>>> as kernel runs on one vector and dosent need many threads
    softmax_kernel<<<1,1>>>(d_output, output_size);

    hipFree(d_hidden);
}

int main() {
    int input_size = 2048;
    int hidden_size = 512;
    int output_size = 10;

    //host allocation
    float *h_input = new float[input_size];
    float *h_W1 = new float[input_size*hidden_size];
    float *h_b1 = new float[hidden_size];
    float *h_W2 = new float[hidden_size*output_size];
    float *h_b2 = new float[output_size];
    float *h_output = new float[output_size];

    //load random data
    for(int i=0;i<input_size;i++) h_input[i] = rand()/(float)RAND_MAX;
    for(int i=0;i<input_size*hidden_size;i++) h_W1[i] = rand()/(float)RAND_MAX;
    for(int i=0;i<hidden_size;i++) h_b1[i] = rand()/(float)RAND_MAX;
    for(int i=0;i<hidden_size*output_size;i++) h_W2[i] = rand()/(float)RAND_MAX;
    for(int i=0;i<output_size;i++) h_b2[i] = rand()/(float)RAND_MAX;

    //device allocation
    float *d_input, *d_W1, *d_b1, *d_W2, *d_b2, *d_output;
    hipMalloc(&d_input, input_size*sizeof(float));
    hipMalloc(&d_W1, input_size*hidden_size*sizeof(float));
    hipMalloc(&d_b1, hidden_size*sizeof(float));
    hipMalloc(&d_W2, hidden_size*output_size*sizeof(float));
    hipMalloc(&d_b2, output_size*sizeof(float));
    hipMalloc(&d_output, output_size*sizeof(float));

    // Copy to GPU
    hipMemcpy(d_input, h_input, input_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W1, h_W1, input_size*hidden_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, h_b1, hidden_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, h_W2, hidden_size*output_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, h_b2, output_size*sizeof(float), hipMemcpyHostToDevice);

    // GPU forward pass produces d_output (raw probabilities after softmax)
    auto start = std::chrono::high_resolution_clock::now();
    forward_gpu(d_input, d_W1, d_b1, d_W2, d_b2, d_output, input_size, hidden_size, output_size);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_output, d_output, output_size*sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "GPU forward pass time: "
              << std::chrono::duration<float, std::milli>(end-start).count() << " ms\n";

    std::cout << "probability of class 0: " << h_output[0] << std::endl;

    // CPU forward pass (naive)
    auto cpu_start = std::chrono::high_resolution_clock::now();
    float hidden[hidden_size];
    for(int i=0;i<hidden_size;i++){
        hidden[i] = h_b1[i];
        for(int j=0;j<input_size;j++) hidden[i] += h_input[j]*h_W1[j*hidden_size + i];
        hidden[i] = fmaxf(0.0f, hidden[i]);
    }
    float output[output_size];
    for(int i=0;i<output_size;i++){
        output[i] = h_b2[i];
        for(int j=0;j<hidden_size;j++) output[i] += hidden[j]*h_W2[j*output_size + i];
    }
    // Softmax
    float max_val = -1e20, sum=0.0f;
    for(int i=0;i<output_size;i++) if(output[i]>max_val) max_val=output[i];
    for(int i=0;i<output_size;i++){
        output[i] = expf(output[i]-max_val);
        sum += output[i];
    }
    for(int i=0;i<output_size;i++) output[i]/=sum;
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::cout << "CPU forward pass time: "
              << std::chrono::duration<float, std::milli>(cpu_end-cpu_start).count() << " ms\n";

    // Free
    hipFree(d_input); hipFree(d_W1); hipFree(d_b1);
    hipFree(d_W2); hipFree(d_b2); hipFree(d_output);
    delete[] h_input; delete[] h_W1; delete[] h_b1;
    delete[] h_W2; delete[] h_b2; delete[] h_output;

    return 0;
}
