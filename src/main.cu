#include <stdio.h>
#include <hip/hip_runtime.h>


#define TILE 16
__global__ void matmul_tiled(const float*, const float*, float*, int, int, int);

int main() {
    const int M = 32, K = 32, N = 32;
    size_t bytesA = M * K * sizeof(float);
    size_t bytesB = K * N * sizeof(float);
    size_t bytesC = M * N * sizeof(float);

    float *h_A = (float*)malloc(bytesA);
    float *h_B = (float*)malloc(bytesB);
    float *h_C = (float*)malloc(bytesC);

    for (int i = 0; i < M * K; i++) h_A[i] = 1.0f;
    for (int i = 0; i < K * N; i++) h_B[i] = 1.0f;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytesA);
    hipMalloc(&d_B, bytesB);
    hipMalloc(&d_C, bytesC);

    hipMemcpy(d_A, h_A, bytesA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytesB, hipMemcpyHostToDevice);

    dim3 threads(TILE, TILE);
    dim3 blocks((N + TILE - 1) / TILE, (M + TILE - 1) / TILE);
    matmul_tiled<<<blocks, threads>>>(d_A, d_B, d_C, M, N, K);
    hipMemcpy(h_C, d_C, bytesC, hipMemcpyDeviceToHost);

    printf("C[0] = %f\n", h_C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
